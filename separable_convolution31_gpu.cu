#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "common.h"

#include "hip/hip_runtime.h"
#include ""


#include "separable_convolution31_gpu.h"

#define LOCAL					static 


LOCAL void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#if(1)
#define getLastCudaError(msg) __getLastCudaError(msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file,
	const int line) {
	hipError_t err = hipGetLastError();

	if (hipSuccess != err) {
		fprintf(stderr,
			"%s(%i) : getLastCudaError() CUDA error :"
			" %s : (%d) %s.\n",
			file, line, errorMessage, static_cast<int>(err),
			hipGetErrorString(err));
		hipDeviceReset();
		exit(EXIT_FAILURE);
	}
}
#endif

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



LOCAL __constant__ float kernel_const_mem[1024];


LOCAL __global__ void SeparateConvolutionColumnGPUKernelInConstSharedMemPadding31CU(
	int width, int height, float const *p_extended_input_dev,
	int kernel_length, float const *p_kernel_column_dev,
	float *p_column_done_extended_output_dev, const int padding)
{
	int i, j;
	int kernel_radius;
	int extended_width;

	extern __shared__ float shared_mem[];
	float *p_input_in_block;
	int block_height;
	int shared_mem_pitch;

	(void)p_kernel_column_dev;

	kernel_radius = kernel_length / 2;
	extended_width = width + 2 * kernel_radius;

	p_input_in_block = &shared_mem[0];

	block_height = kernel_length + (blockDim.y - 1);

	shared_mem_pitch = blockDim.x;
	shared_mem_pitch += padding;

	j = blockDim.y*blockIdx.y + threadIdx.y;
	i = blockDim.x*blockIdx.x + threadIdx.x;

	int jj;
	int x;
	float sum;

	sum = 0;
	x = kernel_radius + i;

#if(0)
	jj = 0;

	do {
		p_input_in_block[(threadIdx.y + jj*blockDim.y)* shared_mem_pitch
			+ threadIdx.x]
			= p_extended_input_dev
			[(j + jj*blockDim.y)*extended_width
			+ kernel_radius + i];

		jj++;
	} while (threadIdx.y + jj * blockDim.y < block_height);
#else
#pragma unroll 3
	for (int jj = 0; jj < 3; jj++) {
		if (threadIdx.y + jj * blockDim.y < block_height) {
			p_input_in_block[(threadIdx.y + jj*blockDim.y)* shared_mem_pitch
				+ threadIdx.x]
				= p_extended_input_dev
				[(j + jj*blockDim.y)*extended_width
				+ kernel_radius + i];
		}
	}
#endif
	__syncthreads();

#pragma unroll KERNEL_LENGTH
	for (jj = 0; jj < KERNEL_LENGTH; jj++) {
		sum += kernel_const_mem[jj] * p_input_in_block[
			(threadIdx.y + jj)*shared_mem_pitch + threadIdx.x];
	}/*for kernel*/

	p_column_done_extended_output_dev[j*extended_width + kernel_radius + i]
		= sum;


}/*SeparateConvolutionColumnGPUKernelInConstSharedMemPadding31CU*/



template<int ii> __device__ int CopyToSharedMemRow(
	int i, int j, 
	int block_width, int extended_width, int shared_mem_pitch,
	float *p_input_in_block, float const *p_column_done_extended_input_dev)
{
	int iii;
	iii = ii - 1;

	if (threadIdx.x + iii * blockDim.x < block_width) {
		p_input_in_block[threadIdx.y*shared_mem_pitch
			+ iii*blockDim.x + threadIdx.x] =
			p_column_done_extended_input_dev[j*extended_width
			+ iii*blockDim.x + i];
	}

	CopyToSharedMemRow<ii - 1>(i, j,
		block_width, extended_width, shared_mem_pitch,
		p_input_in_block, p_column_done_extended_input_dev);
}

template<> __device__ int CopyToSharedMemRow<0>(
	int i, int j,
	int block_width, int extended_width, int shared_mem_pitch,
	float *p_input_in_block ,float const *p_column_done_extended_input_dev)
{
	return 0;
}

LOCAL __global__ void SeparateConvolutionRowGPUKernelInConstSharedMemPadding31CU(
	int width, int height, float const *p_column_done_extended_input_dev,
	int kernel_length, float const *p_kernel_row_dev,
	float *p_output_dev, const int padding)
{
	int i, j;
	int kernel_radius;
	int extended_width;

	extern __shared__ float shared_mem[];
	float *p_input_in_block;
	int block_width;
	int shared_mem_pitch;


	(void)p_kernel_row_dev;

	kernel_radius = kernel_length / 2;
	extended_width = width + 2 * kernel_radius;

	p_input_in_block = &shared_mem[0];
	block_width = kernel_length + (2*blockDim.x - 1);

	shared_mem_pitch = block_width;	
	shared_mem_pitch += padding;


	j = 2 * blockDim.y*blockIdx.y + threadIdx.y;
	i = 2 * blockDim.x*blockIdx.x + threadIdx.x;	
	

	int ii;
	float sum;

	sum = 0;
	
#if(1)
	ii = 0;
	do {
		p_input_in_block[threadIdx.y*shared_mem_pitch
			+ ii*blockDim.x + threadIdx.x] =
			p_column_done_extended_input_dev[j*extended_width
			+ ii*blockDim.x + i];		

		ii++;
	} while (threadIdx.x + ii * blockDim.x < block_width);
	
	ii = 0;
	do {
		p_input_in_block[(threadIdx.y + blockDim.y)*shared_mem_pitch
			+ ii*blockDim.x + threadIdx.x] =
			p_column_done_extended_input_dev[(j + blockDim.y)*extended_width
			+ ii*blockDim.x + i];
		ii++;
	} while (threadIdx.x + ii * blockDim.x < block_width);
	
#else
#if(1)
#pragma unroll 3
	for (int ii = 0; ii < 3; ii++) {
		if (threadIdx.x + ii * blockDim.x < block_width) {
			p_input_in_block[threadIdx.y*shared_mem_pitch
				+ ii*blockDim.x + threadIdx.x] =
				p_column_done_extended_input_dev[j*extended_width
				+ ii*blockDim.x + i];
		}
	}

#else
	CopyToSharedMemRow<3>(i, j, 
		block_width, extended_width, shared_mem_pitch,
		p_input_in_block, p_column_done_extended_input_dev);
	

#endif
#endif
	__syncthreads();

	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (ii = 0; ii < KERNEL_LENGTH; ii++) {
		sum += kernel_const_mem[ii] * p_input_in_block[
			threadIdx.y*shared_mem_pitch + ii + threadIdx.x];
	}/*for kernel_length*/

	p_output_dev[j*width + i] = sum;

	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (ii = 0; ii < KERNEL_LENGTH; ii++) {
		sum += kernel_const_mem[ii] * p_input_in_block[
			threadIdx.y*shared_mem_pitch + ii + threadIdx.x + blockDim.x];
	}/*for kernel_length*/

	p_output_dev[j*width + i + blockDim.x] = sum;

	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (ii = 0; ii < KERNEL_LENGTH; ii++) {
		sum += kernel_const_mem[ii] * p_input_in_block[
			(threadIdx.y + blockDim.y)*shared_mem_pitch + ii + threadIdx.x];
	}/*for kernel_length*/

	p_output_dev[(j + blockDim.y)*width + i] = sum;

	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (ii = 0; ii < KERNEL_LENGTH; ii++) {
		sum += kernel_const_mem[ii] * p_input_in_block[
			(threadIdx.y + blockDim.y)*shared_mem_pitch + ii + threadIdx.x + blockDim.x];
	}/*for kernel_length*/

	
	p_output_dev[(j + blockDim.y)* width + i + blockDim.x] = sum;
}/*SeparateConvolutionRowGPUKernelInConstSharedMemPadding31CU*/

#define WARP_SIZE					(32)

int SeparableConvolutionColumnGPUKernelInConstSharedMemPadding31(
	dim3 num_blocks, dim3 num_threads,
	int width, int height, float const *p_extended_input_dev,
	int kernel_length, float const *p_kernel_column_host,
	float *p_column_done_extended_output_dev)
{
	int extended_width;
	float *p_kernel_const_dev;
	int shared_mem_size;
	int kernel_radius;
	
	int block_height;
	int padding;


	if (0 == width || 0 == height)
		return -1;

	if (kernel_length > width || kernel_length > height)
		return -2;

	kernel_radius = kernel_length / 2;
	extended_width = width + 2 * kernel_radius;

	block_height = kernel_length + (num_threads.y - 1);

/*
	padding
	= WARP_SIZE*n - (block_size + num_threads + (WARP_SIZE - num_threads))
*/

/*
	padding = num_threads.x + (WARP_SIZE - num_threads.x);
*/
	padding = 0;
	shared_mem_size = sizeof(float)
		* (num_threads.x + padding) *(block_height);	

	HANDLE_ERROR(hipGetSymbolAddress((void **)&p_kernel_const_dev,
		kernel_const_mem));

	HANDLE_ERROR(hipMemcpy(p_kernel_const_dev, p_kernel_column_host,
		kernel_length * sizeof(float), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemset(p_column_done_extended_output_dev, 0,
		extended_width*height * sizeof(float)));

	SeparateConvolutionColumnGPUKernelInConstSharedMemPadding31CU
		<< <num_blocks, num_threads, shared_mem_size >> >
		(width, height, p_extended_input_dev, kernel_length,
			NULL, p_column_done_extended_output_dev, padding);

	getLastCudaError("SeparateConvolutionColumnGPUKernelInConstSharedMemPadding31CU");
	return 0;
}/*SeparableConvolutionColumnGPUKernelInConstSharedMemPadding*/


int SeparableConvolutionRowGPUKernelInConstSharedMemPadding31(
	dim3 num_blocks, dim3 num_threads,
	int width, int height, float const *p_column_done_extended_input_dev,
	int kernel_length, float const *p_kernel_row_host,
	float *p_output_dev)
{
	float *p_kernel_const_dev;
	int shared_mem_size;
	int kernel_radius;

	int block_width;	
	int padding;

	if (0 == width || 0 == height)
		return -1;

	if (kernel_length > width || kernel_length > height)
		return -2;

	kernel_radius = kernel_length / 2;
	block_width = kernel_length + (num_threads.x - 1);
	block_width += num_threads.x;
	//int n;
	//n = (block_width + (num_threads.x - 1)) / num_threads.x;

	//block_width = num_threads.x * n;
/*
	padding 
	= WARP_SIZE*n - (block_size + (WARP_SIZE - num_threads))
*/
	{
		int temp = block_width + (WARP_SIZE - num_threads.x);

		padding = WARP_SIZE*((temp + (WARP_SIZE - 1)) / WARP_SIZE)
			- temp;
	}/*local variable*/

	shared_mem_size = sizeof(float) 
		* (block_width + padding) *(2*num_threads.y);
	
	HANDLE_ERROR(hipGetSymbolAddress((void **)&p_kernel_const_dev,
		kernel_const_mem));

	HANDLE_ERROR(hipMemcpy(p_kernel_const_dev, p_kernel_row_host,
		kernel_length * sizeof(float), hipMemcpyHostToDevice));

	num_blocks.x /= 2;
	num_blocks.y /= 2;

	SeparateConvolutionRowGPUKernelInConstSharedMemPadding31CU
		<< <num_blocks, num_threads, shared_mem_size >> >
		(width, height, p_column_done_extended_input_dev,
			kernel_length, NULL, p_output_dev, padding);

	getLastCudaError("SeparateConvolutionRowGPUKernelInConstSharedMemPadding31CU");

	return 0;
}/*SeparableConvolutionRowGPUKernelInConstSharedMemPadding31*/
