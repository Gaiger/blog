#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "common.h"

#include "hip/hip_runtime.h"
#include ""


#include "separable_convolution_31_1400x1400_gpu.h"

#define LOCAL					static 


LOCAL void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#if(1)
#define getLastCudaError(msg) __getLastCudaError(msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file,
	const int line) {
	hipError_t err = hipGetLastError();

	if (hipSuccess != err) {
		fprintf(stderr,
			"%s(%i) : getLastCudaError() CUDA error :"
			" %s : (%d) %s.\n",
			file, line, errorMessage, static_cast<int>(err),
			hipGetErrorString(err));
		hipDeviceReset();
		exit(EXIT_FAILURE);
	}
}
#endif

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



LOCAL __constant__ float kernel_const_mem[1024];

#define COLUMN_CP_STEPS						\
	(KERNEL_LENGTH + 2*Y_NUM_THREADS - 1 + (Y_NUM_THREADS - 1))/(Y_NUM_THREADS)


template<int jj> __device__ int CopyToSharedMemColumn(
	int i, int j,
	int block_height, int extended_width, int shared_mem_pitch,
	int kernel_radius,
	float *p_input_in_block, float const *p_extended_input_dev)
{
	int jjj;
	jjj = COLUMN_CP_STEPS - jj;

	p_input_in_block[(threadIdx.y + jjj*blockDim.y)* shared_mem_pitch
		+ threadIdx.x]
		= p_extended_input_dev
		[(j + jjj*blockDim.y)*extended_width
		+ kernel_radius + i];

	p_input_in_block[(threadIdx.y + jjj*blockDim.y)* shared_mem_pitch
		+ threadIdx.x + blockDim.x]
		= p_extended_input_dev
		[(j + jjj*blockDim.y)*extended_width
		+ kernel_radius + i + blockDim.x];


	CopyToSharedMemColumn<jj - 1>(i, j,
		block_height, extended_width, shared_mem_pitch, kernel_radius,
		p_input_in_block, p_extended_input_dev);
}/*CopyToSharedMemRow*/

template<> __device__ int CopyToSharedMemColumn<1>(
	int i, int j,
	int block_height, int extended_width, int shared_mem_pitch,
	int kernel_radius,
	float *p_input_in_block, float const *p_extended_input_dev)
{
	int jjj;
	jjj = COLUMN_CP_STEPS - 1;
	if (threadIdx.y + jjj * blockDim.y < block_height) {
		p_input_in_block[(threadIdx.y + jjj*blockDim.y)* shared_mem_pitch
			+ threadIdx.x]
			= p_extended_input_dev
			[(j + jjj*blockDim.y)*extended_width
			+ kernel_radius + i];

		p_input_in_block[(threadIdx.y + jjj*blockDim.y)* shared_mem_pitch
			+ threadIdx.x + blockDim.x]
			= p_extended_input_dev
			[(j + jjj*blockDim.y)*extended_width
			+ kernel_radius + i + blockDim.x];
	}/*if */

	return 0;
}/*CopyToSharedMemRow<1>*/


LOCAL __global__ void SeparateConvolutionColumnGPU_31_1400x1400CU(
	int width, int height, float const *p_extended_input_dev,
	int kernel_length, float const *p_kernel_column_dev,
	float *p_column_done_extended_output_dev, const int padding)
{
	int i, j;
	int kernel_radius;
	int extended_width;

	extern __shared__ float shared_mem[];
	float *p_input_in_block;
	int block_height;
	int shared_mem_pitch;

	(void)p_kernel_column_dev;

	kernel_radius = kernel_length / 2;
	extended_width = width + 2 * kernel_radius;

	p_input_in_block = &shared_mem[0];

	block_height = kernel_length + ( 2*blockDim.y - 1);

	shared_mem_pitch = 2 * blockDim.x;
	shared_mem_pitch += padding;

	j = 2 * blockDim.y*blockIdx.y + threadIdx.y;
	i = 2 * blockDim.x*blockIdx.x + threadIdx.x;

	int jj;
	float sum;

#if(0)
	jj = 0;

	do {
		p_input_in_block[(threadIdx.y + jj*blockDim.y)* shared_mem_pitch
			+ threadIdx.x]
			= p_extended_input_dev
			[(j + jj*blockDim.y)*extended_width
			+ kernel_radius + i];

		p_input_in_block[(threadIdx.y + jj*blockDim.y)* shared_mem_pitch
			+ threadIdx.x + blockDim.x]
			= p_extended_input_dev
			[(j + jj*blockDim.y)*extended_width
			+ kernel_radius + i + blockDim.x];
		jj++;
	} while (threadIdx.y + jj * blockDim.y < block_height);
	
#else
#if(0)
#pragma unroll (COLUMN_CP_STEPS - 1)
	for (int jj = 0; jj < (COLUMN_CP_STEPS - 1) ; jj++) {
			p_input_in_block[(threadIdx.y + jj*blockDim.y)* shared_mem_pitch
				+ threadIdx.x]
				= p_extended_input_dev
				[(j + jj*blockDim.y)*extended_width
				+ kernel_radius + i];

			p_input_in_block[(threadIdx.y + jj*blockDim.y)* shared_mem_pitch
				+ threadIdx.x + blockDim.x]
				= p_extended_input_dev
				[(j + jj*blockDim.y)*extended_width
				+ kernel_radius + i + blockDim.x];		
	}/*for */

	jj = (COLUMN_CP_STEPS - 1);
	if (threadIdx.y + jj * blockDim.y < block_height) {
		p_input_in_block[(threadIdx.y + jj*blockDim.y)* shared_mem_pitch
			+ threadIdx.x]
			= p_extended_input_dev
			[(j + jj*blockDim.y)*extended_width
			+ kernel_radius + i];

		p_input_in_block[(threadIdx.y + jj*blockDim.y)* shared_mem_pitch
			+ threadIdx.x + blockDim.x]
			= p_extended_input_dev
			[(j + jj*blockDim.y)*extended_width
			+ kernel_radius + i + blockDim.x];
	}
#else
	CopyToSharedMemColumn<COLUMN_CP_STEPS>(i, j, block_height, extended_width, shared_mem_pitch,
		kernel_radius, p_input_in_block, p_extended_input_dev);
#endif
#endif
	__syncthreads();

	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (jj = 0; jj < KERNEL_LENGTH; jj++) {
		sum += kernel_const_mem[jj] * p_input_in_block[
			(threadIdx.y + jj)*shared_mem_pitch + threadIdx.x];
	}/*for kernel*/

	p_column_done_extended_output_dev[j*extended_width + kernel_radius + i]
		= sum;


	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (jj = 0; jj < KERNEL_LENGTH; jj++) {
		sum += kernel_const_mem[jj] * p_input_in_block[
			(threadIdx.y + jj)*shared_mem_pitch + threadIdx.x + blockDim.x];
	}/*for kernel*/

	p_column_done_extended_output_dev[j*extended_width + kernel_radius + i + blockDim.x]
		= sum;


	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (jj = 0; jj < KERNEL_LENGTH; jj++) {
		sum += kernel_const_mem[jj] * p_input_in_block[
			(threadIdx.y + jj + blockDim.y)*shared_mem_pitch + threadIdx.x];
	}/*for kernel*/

	p_column_done_extended_output_dev[(j + blockDim.y)*extended_width + kernel_radius + i]
		= sum;


	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (jj = 0; jj < KERNEL_LENGTH; jj++) {
		sum += kernel_const_mem[jj] * p_input_in_block[
			(threadIdx.y + jj + blockDim.y)*shared_mem_pitch + threadIdx.x + blockDim.x];
	}/*for kernel*/

	p_column_done_extended_output_dev[(j + blockDim.y)*extended_width + kernel_radius + i + blockDim.x]
		= sum;

}/*SeparateConvolutionColumnGPU_31_1400x1400CU*/

#define ROW_CP_STEPS						\
	(KERNEL_LENGTH + 2*X_NUM_THREADS - 1 + (X_NUM_THREADS - 1))/(X_NUM_THREADS)


template<int ii> __device__ int CopyToSharedMemRow(
	int i, int j,
	int block_width, int extended_width, int shared_mem_pitch,
	float *p_input_in_block, float const *p_column_done_extended_input_dev)
{
	int iii;

	iii = ROW_CP_STEPS - ii;

	CopyToSharedMemRow<ii - 1>(i, j,
		block_width, extended_width, shared_mem_pitch,
		p_input_in_block, p_column_done_extended_input_dev);

	p_input_in_block[threadIdx.y*shared_mem_pitch
		+ iii*blockDim.x + threadIdx.x] =
		p_column_done_extended_input_dev[j*extended_width
		+ iii*blockDim.x + i];

	p_input_in_block[(threadIdx.y + blockDim.y)*shared_mem_pitch
		+ iii*blockDim.x + threadIdx.x] =
		p_column_done_extended_input_dev[(j + blockDim.y)*extended_width
		+ iii*blockDim.x + i];

}/*CopyToSharedMemRow*/

template<> __device__ int CopyToSharedMemRow<1>(
	int i, int j,
	int block_width, int extended_width, int shared_mem_pitch,
	float *p_input_in_block, float const *p_column_done_extended_input_dev)
{
	int iii = ROW_CP_STEPS - 1;

	if (threadIdx.x + iii * blockDim.x < block_width) {
		p_input_in_block[threadIdx.y*shared_mem_pitch
			+ iii*blockDim.x + threadIdx.x] =
			p_column_done_extended_input_dev[j*extended_width
			+ iii*blockDim.x + i];

		p_input_in_block[(threadIdx.y + blockDim.y)*shared_mem_pitch
			+ iii*blockDim.x + threadIdx.x] =
			p_column_done_extended_input_dev[(j + blockDim.y)*extended_width
			+ iii*blockDim.x + i];
	}/*if */

	return 0;
}/*CopyToSharedMemRow<1>*/


LOCAL __global__ void SeparateConvolutionRowGPU_31_1400x1400CU(
	int width, int height, float const *p_column_done_extended_input_dev,
	int kernel_length, float const *p_kernel_row_dev,
	float *p_output_dev, const int padding)
{
	int i, j;
	int kernel_radius;
	int extended_width;

	extern __shared__ float shared_mem[];
	float *p_input_in_block;
	int block_width;
	int shared_mem_pitch;


	(void)p_kernel_row_dev;

	kernel_radius = kernel_length / 2;
	extended_width = width + 2 * kernel_radius;

	p_input_in_block = &shared_mem[0];
	block_width = kernel_length + (2 * blockDim.x - 1);

	shared_mem_pitch = block_width;	
	shared_mem_pitch += padding;


	j = 2 * blockDim.y*blockIdx.y + threadIdx.y;
	i = 2 * blockDim.x*blockIdx.x + threadIdx.x;	
	

	int ii;
	float sum;

	sum = 0;
	
#if(0)
	ii = 0;
	do {
		p_input_in_block[threadIdx.y*shared_mem_pitch
			+ ii*blockDim.x + threadIdx.x] =
			p_column_done_extended_input_dev[j*extended_width
			+ ii*blockDim.x + i];		
		
		p_input_in_block[(threadIdx.y + blockDim.y)*shared_mem_pitch
			+ ii*blockDim.x + threadIdx.x] =
			p_column_done_extended_input_dev[(j + blockDim.y)*extended_width
			+ ii*blockDim.x + i];

		ii++;
	} while (threadIdx.x + ii * blockDim.x < block_width);
	
#else


#if(1)

#pragma unroll (ROW_CP_STEPS - 1)
	for (int ii = 0; ii < (ROW_CP_STEPS - 1); ii++) {
		if (threadIdx.x + ii * blockDim.x < block_width) {
			p_input_in_block[threadIdx.y*shared_mem_pitch
				+ ii*blockDim.x + threadIdx.x] =
				p_column_done_extended_input_dev[j*extended_width
				+ ii*blockDim.x + i];

			p_input_in_block[(threadIdx.y + blockDim.y)*shared_mem_pitch
				+ ii*blockDim.x + threadIdx.x] =
				p_column_done_extended_input_dev[(j + blockDim.y)*extended_width
				+ ii*blockDim.x + i];
		}
	}/*for */

	ii = ROW_CP_STEPS - 1;
	if (threadIdx.x + ii * blockDim.x < block_width) {
		p_input_in_block[threadIdx.y*shared_mem_pitch
			+ ii*blockDim.x + threadIdx.x] =
			p_column_done_extended_input_dev[j*extended_width
			+ ii*blockDim.x + i];

		p_input_in_block[(threadIdx.y + blockDim.y)*shared_mem_pitch
			+ ii*blockDim.x + threadIdx.x] =
			p_column_done_extended_input_dev[(j + blockDim.y)*extended_width
			+ ii*blockDim.x + i];
	}

#else
	CopyToSharedMemRow<ROW_CP_STEPS>(i, j,
		block_width, extended_width, shared_mem_pitch,
		p_input_in_block, p_column_done_extended_input_dev);
	

#endif
#endif
	__syncthreads();

	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (ii = 0; ii < KERNEL_LENGTH; ii++) {
		sum += kernel_const_mem[ii] * p_input_in_block[
			threadIdx.y*shared_mem_pitch + ii + threadIdx.x];
	}/*for kernel_length*/

	p_output_dev[j*width + i] = sum;

	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (ii = 0; ii < KERNEL_LENGTH; ii++) {
		sum += kernel_const_mem[ii] * p_input_in_block[
			threadIdx.y*shared_mem_pitch + ii + threadIdx.x + blockDim.x];
	}/*for kernel_length*/

	p_output_dev[j*width + i + blockDim.x] = sum;

	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (ii = 0; ii < KERNEL_LENGTH; ii++) {
		sum += kernel_const_mem[ii] * p_input_in_block[
			(threadIdx.y + blockDim.y)*shared_mem_pitch + ii + threadIdx.x];
	}/*for kernel_length*/

	p_output_dev[(j + blockDim.y)*width + i] = sum;

	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (ii = 0; ii < KERNEL_LENGTH; ii++) {
		sum += kernel_const_mem[ii] * p_input_in_block[
			(threadIdx.y + blockDim.y)*shared_mem_pitch + ii + threadIdx.x + blockDim.x];
	}/*for kernel_length*/

	
	p_output_dev[(j + blockDim.y)* width + i + blockDim.x] = sum;
}/*SeparateConvolutionRowGPU_31_1400x1400CU*/

#define WARP_SIZE					(32)

int SeparableConvolutionColumnGPU_31_1400x1400(
	dim3 num_blocks, dim3 num_threads,
	int width, int height, float const *p_extended_input_dev,
	int kernel_length, float const *p_kernel_column_host,
	float *p_column_done_extended_output_dev)
{
	int extended_width;
	float *p_kernel_const_dev;
	int shared_mem_size;
	int kernel_radius;
	
	int block_height;
	int padding;


	if (0 == width || 0 == height)
		return -1;

	if (kernel_length > width || kernel_length > height)
		return -2;

	kernel_radius = kernel_length / 2;
	extended_width = width + 2 * kernel_radius;

	block_height = kernel_length + (num_threads.y - 1);
	block_height += num_threads.y;

/*
	padding
	= WARP_SIZE*n - (block_size + num_threads + (WARP_SIZE - num_threads))
*/

/*
	padding = num_threads.x + (WARP_SIZE - num_threads.x);
*/
	padding = 0;
	shared_mem_size = sizeof(float)
		* ( 2 * num_threads.x + padding) *( block_height);	

	HANDLE_ERROR(hipGetSymbolAddress((void **)&p_kernel_const_dev,
		kernel_const_mem));

	HANDLE_ERROR(hipMemcpy(p_kernel_const_dev, p_kernel_column_host,
		kernel_length * sizeof(float), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemset(p_column_done_extended_output_dev, 0,
		extended_width*height * sizeof(float)));

	num_blocks.x /= 2;
	num_blocks.y /= 2;
	
	SeparateConvolutionColumnGPU_31_1400x1400CU
		<< <num_blocks, num_threads, shared_mem_size >> >
		(width, height, p_extended_input_dev, kernel_length,
			NULL, p_column_done_extended_output_dev, padding);

	getLastCudaError("SeparateConvolutionColumnGPU_31_1400x1400CU");
	return 0;
}/*SeparableConvolutionColumnGPU_31_1400x1400*/


int SeparableConvolutionRowGPU_31_1400x1400(
	dim3 num_blocks, dim3 num_threads,
	int width, int height, float const *p_column_done_extended_input_dev,
	int kernel_length, float const *p_kernel_row_host,
	float *p_output_dev)
{
	float *p_kernel_const_dev;
	int shared_mem_size;
	int kernel_radius;

	int block_width;	
	int padding;

	if (0 == width || 0 == height)
		return -1;

	if (kernel_length > width || kernel_length > height)
		return -2;

	kernel_radius = kernel_length / 2;
	block_width = kernel_length + (num_threads.x - 1);
	block_width += num_threads.x;
	//int n;
	//n = (block_width + (num_threads.x - 1)) / num_threads.x;

	//block_width = num_threads.x * n;
/*
	padding 
	= WARP_SIZE*n - (block_size + (WARP_SIZE - num_threads))
*/
	{
		int temp = block_width + (WARP_SIZE - num_threads.x);

		padding = WARP_SIZE*((temp + (WARP_SIZE - 1)) / WARP_SIZE)
			- temp;
	}/*local variable*/

	shared_mem_size = sizeof(float) 
		* (block_width + padding) *(2*num_threads.y);
	
	HANDLE_ERROR(hipGetSymbolAddress((void **)&p_kernel_const_dev,
		kernel_const_mem));

	HANDLE_ERROR(hipMemcpy(p_kernel_const_dev, p_kernel_row_host,
		kernel_length * sizeof(float), hipMemcpyHostToDevice));

	num_blocks.x /= 2;
	num_blocks.y /= 2;

	SeparateConvolutionRowGPU_31_1400x1400CU
		<< <num_blocks, num_threads, shared_mem_size >> >
		(width, height, p_column_done_extended_input_dev,
			kernel_length, NULL, p_output_dev, padding);

	getLastCudaError("SeparableConvolutionRowGPU_31_1400x1400");

	return 0;
}/*SeparableConvolutionRowGPU_31_1400x1400*/
