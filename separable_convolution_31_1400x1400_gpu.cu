#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "common.h"

#include "hip/hip_runtime.h"
#include ""


#include "separable_convolution_31_1400x1400_gpu.h"

#define LOCAL					static 


LOCAL void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#if(1)
#define getLastCudaError(msg) __getLastCudaError(msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file,
	const int line) {
	hipError_t err = hipGetLastError();

	if (hipSuccess != err) {
		fprintf(stderr,
			"%s(%i) : getLastCudaError() CUDA error :"
			" %s : (%d) %s.\n",
			file, line, errorMessage, static_cast<int>(err),
			hipGetErrorString(err));
		hipDeviceReset();
		exit(EXIT_FAILURE);
	}
}
#endif

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define COLUMN_CP_STEPS						\
	(KERNEL_LENGTH + 2*Y_NUM_THREADS - 1 + (Y_NUM_THREADS - 1))/(Y_NUM_THREADS)

#define ROW_CP_STEPS						\
	(KERNEL_LENGTH + 2*X_NUM_THREADS - 1 + (X_NUM_THREADS - 1))/(X_NUM_THREADS)


LOCAL __constant__ float kernel_const_mem[1024];


LOCAL __global__ void SeparateConvolutionColumnGPU_31_1400x1400_UnrollCU(
	int width, int height, float const *p_extended_input_dev,
	int kernel_length, float const *p_kernel_column_dev,
	float *p_column_done_extended_output_dev, const int padding)
{
	int i, j;
	int kernel_radius;
	int extended_width;

	extern __shared__ float shared_mem[];
	float *p_input_in_block;
	int block_height;
	int shared_mem_pitch;

	(void)p_kernel_column_dev;

	kernel_radius = kernel_length / 2;
	extended_width = width + 2 * kernel_radius;

	p_input_in_block = &shared_mem[0];

	block_height = kernel_length + (blockDim.y - 1);

	shared_mem_pitch = blockDim.x;
	shared_mem_pitch += padding;

	j = blockDim.y*blockIdx.y + threadIdx.y;
	i = blockDim.x*blockIdx.x + threadIdx.x;

	int jj;
	float sum;

#if(0)
	jj = 0;

	do {
		p_input_in_block[(threadIdx.y + jj*blockDim.y)* shared_mem_pitch
			+ threadIdx.x]
			= p_extended_input_dev
			[(j + jj*blockDim.y)*extended_width
			+ kernel_radius + i];
		jj++;
	} while (threadIdx.y + jj * blockDim.y < block_height);
	
#else
#pragma unroll (COLUMN_CP_STEPS - 1)
	for (int jj = 0; jj < (COLUMN_CP_STEPS - 1) ; jj++ ) {
			p_input_in_block[(threadIdx.y + jj*blockDim.y)* shared_mem_pitch
				+ threadIdx.x]
				= p_extended_input_dev
				[(j + jj*blockDim.y)*extended_width
				+ kernel_radius + i];
	}/*for */

	jj = (COLUMN_CP_STEPS - 1);
	if (threadIdx.y + jj * blockDim.y < block_height) {
		p_input_in_block[(threadIdx.y + jj*blockDim.y)* shared_mem_pitch
			+ threadIdx.x]
			= p_extended_input_dev
			[(j + jj*blockDim.y)*extended_width
			+ kernel_radius + i];
	}/*COLUMN_CP_STEPS - 1*/
#endif

	__syncthreads();

	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (jj = 0; jj < KERNEL_LENGTH; jj++) {
		sum += kernel_const_mem[jj] * p_input_in_block[
			(threadIdx.y + jj)*shared_mem_pitch + threadIdx.x];
	}/*for kernel*/

	p_column_done_extended_output_dev[j*extended_width + kernel_radius + i]
		= sum;

}/*SeparateConvolutionColumnGPU_31_1400x1400_UnrollCU*/


LOCAL __global__ void SeparateConvolutionRowGPU_31_1400x1400_UnrollCU(
	int width, int height, float const *p_column_done_extended_input_dev,
	int kernel_length, float const *p_kernel_row_dev,
	float *p_output_dev, const int padding)
{
	int i, j;
	int kernel_radius;
	int extended_width;

	extern __shared__ float shared_mem[];
	float *p_input_in_block;
	int block_width;
	int shared_mem_pitch;


	(void)p_kernel_row_dev;

	kernel_radius = kernel_length / 2;
	extended_width = width + 2 * kernel_radius;

	p_input_in_block = &shared_mem[0];
	block_width = kernel_length + (blockDim.x - 1);

	shared_mem_pitch = block_width;	
	shared_mem_pitch += padding;


	j = blockDim.y*blockIdx.y + threadIdx.y;
	i = blockDim.x*blockIdx.x + threadIdx.x;	
	

	int ii;
	float sum;
	
#if(0)
	ii = 0;
	do {
		p_input_in_block[threadIdx.y*shared_mem_pitch
			+ ii*blockDim.x + threadIdx.x] =
			p_column_done_extended_input_dev[j*extended_width
			+ ii*blockDim.x + i];		

		ii++;
	} while (threadIdx.x + ii * blockDim.x < block_width);
	
#else
#pragma unroll (ROW_CP_STEPS - 1)
	for (int ii = 0; ii < (ROW_CP_STEPS - 1); ii++ ) {
		if (threadIdx.x + ii * blockDim.x < block_width) {
			p_input_in_block[threadIdx.y*shared_mem_pitch
				+ ii*blockDim.x + threadIdx.x] =
				p_column_done_extended_input_dev[j*extended_width
				+ ii*blockDim.x + i];
		}
	}/*for */

	ii = ROW_CP_STEPS - 1;
	if (threadIdx.x + ii * blockDim.x < block_width) {
		p_input_in_block[threadIdx.y*shared_mem_pitch
			+ ii*blockDim.x + threadIdx.x] =
			p_column_done_extended_input_dev[j*extended_width
			+ ii*blockDim.x + i];
	}/* ROW_CP_STEPS - 1*/
#endif
	__syncthreads();

	sum = 0;
#pragma unroll KERNEL_LENGTH
	for (ii = 0; ii < KERNEL_LENGTH; ii++) {
		sum += kernel_const_mem[ii] * p_input_in_block[
			threadIdx.y*shared_mem_pitch + ii + threadIdx.x];
	}/*for kernel_length*/

	p_output_dev[j*width + i] = sum;

}/*SeparateConvolutionRowGPU_31_1400x1400_UnrollCU*/

#define WARP_SIZE					(32)

int SeparableConvolutionColumnGPU_31_1400x1400_Unroll(
	dim3 num_blocks, dim3 num_threads,
	int width, int height, float const *p_extended_input_dev,
	int kernel_length, float const *p_kernel_column_host,
	float *p_column_done_extended_output_dev)
{
	int extended_width;
	float *p_kernel_const_dev;
	int shared_mem_size;
	int kernel_radius;
	
	int block_height;
	int padding;


	if (0 == width || 0 == height)
		return -1;

	if (kernel_length > width || kernel_length > height)
		return -2;

	kernel_radius = kernel_length / 2;
	extended_width = width + 2 * kernel_radius;

	block_height = kernel_length + (num_threads.y - 1);
	block_height += num_threads.y;

/*
	padding
	= WARP_SIZE*n - (block_size + num_threads + (WARP_SIZE - num_threads))
*/

/*
	padding = num_threads.x + (WARP_SIZE - num_threads.x);
*/
	padding = 0;
	shared_mem_size = sizeof(float)
		* (num_threads.x + padding) * (block_height);	

	HANDLE_ERROR(hipGetSymbolAddress((void **)&p_kernel_const_dev,
		kernel_const_mem));

	HANDLE_ERROR(hipMemcpy(p_kernel_const_dev, p_kernel_column_host,
		kernel_length * sizeof(float), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemset(p_column_done_extended_output_dev, 0,
		extended_width*height * sizeof(float)));
	
	SeparateConvolutionColumnGPU_31_1400x1400_UnrollCU
		<< <num_blocks, num_threads, shared_mem_size >> >
		(width, height, p_extended_input_dev, kernel_length,
			NULL, p_column_done_extended_output_dev, padding);

	getLastCudaError("SeparateConvolutionColumnGPU_31_1400x1400_UnrollCU");
	return 0;
}/*SeparableConvolutionColumnGPU_31_1400x1400_Unroll*/


int SeparableConvolutionRowGPU_31_1400x1400_Unroll(
	dim3 num_blocks, dim3 num_threads,
	int width, int height, float const *p_column_done_extended_input_dev,
	int kernel_length, float const *p_kernel_row_host,
	float *p_output_dev)
{
	float *p_kernel_const_dev;
	int shared_mem_size;
	int kernel_radius;

	int block_width;	
	int padding;

	if (0 == width || 0 == height)
		return -1;

	if (kernel_length > width || kernel_length > height)
		return -2;

	kernel_radius = kernel_length / 2;
	block_width = kernel_length + (num_threads.x - 1);

	//int n;
	//n = (block_width + (num_threads.x - 1)) / num_threads.x;

	//block_width = num_threads.x * n;
/*
	padding 
	= WARP_SIZE*n - (block_size + (WARP_SIZE - num_threads))
*/
	{
		int temp = block_width + (WARP_SIZE - num_threads.x);

		padding = WARP_SIZE*((temp + (WARP_SIZE - 1)) / WARP_SIZE)
			- temp;
	}/*local variable*/

	shared_mem_size = sizeof(float) 
		* (block_width + padding ) *(num_threads.y);
	
	HANDLE_ERROR(hipGetSymbolAddress((void **)&p_kernel_const_dev,
		kernel_const_mem));

	HANDLE_ERROR(hipMemcpy(p_kernel_const_dev, p_kernel_row_host,
		kernel_length * sizeof(float), hipMemcpyHostToDevice));


	SeparateConvolutionRowGPU_31_1400x1400_UnrollCU
		<< <num_blocks, num_threads, shared_mem_size >> >
		(width, height, p_column_done_extended_input_dev,
			kernel_length, NULL, p_output_dev, padding);

	getLastCudaError("SeparateConvolutionRowGPU_31_1400x1400_UnrollCU");

	return 0;
}/*SeparableConvolutionRowGPU_31_1400x1400_Unrolll*/
